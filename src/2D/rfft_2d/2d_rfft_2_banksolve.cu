#include "hip/hip_runtime.h"
#include <mma.h>

#include "../create_fft_pfa_plan.cu"
#include "../elementwise_mul.cuh"

#define WMMA_M 8
#define WMMA_N 8
#define WMMA_K 4

#define WARP_SIZE 32

using namespace nvcuda;

template <int N_WARP>
__global__ void rfft_2d_8_nwarp(const double *__restrict__ input,

                                const int ACTUAL_WIDTH,

                                const int INPUT_WIDTH,

                                const int sub_input_width,

                                const int OVERLAP_SIZE,

                                double *output)
{
    extern __shared__ double sharedmem[];

    const int warp_id = threadIdx.x / WARP_SIZE; // TODO: 1

    double *real_shared = &sharedmem[0 + warp_id * shared_unit];
    double *imag_shared = &sharedmem[shared_unit * N_WARP + warp_id * shared_unit];

    const int laneId = threadIdx.x & 0x1f;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, double, wmma::row_major> a_frag_real[2];
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, double, wmma::row_major> a_frag_imag[2];

    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, double, wmma::row_major> b_frag_real[2];
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, double, wmma::row_major> b_frag_imag[2];

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, double> fft_real;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, double> fft_imag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, double> k_real;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, double> k_imag;

    const int intput_idx_x = blockIdx.y * unit;
    const int intput_idx_y = blockIdx.x * 2 * unit;
    const int input_idx_real = intput_idx_x * ACTUAL_WIDTH + intput_idx_y;
    const int input_idx_imag = input_idx_real + unit;

    /**
     * @brief f f t
     *
     */
    // 8 * 8 @ 8 * 8（7）
    // dht_matrix_row @ real_shared
    wmma::fill_fragment(k_real, 0.0);
    wmma::fill_fragment(k_imag, 0.0);
#pragma unroll
    for (size_t i = 0; i < 2; i++)
    {
        wmma::load_matrix_sync(b_frag_imag[i], input + input_idx_imag + i * WMMA_K * ACTUAL_WIDTH, ACTUAL_WIDTH); // d

        wmma::load_matrix_sync(a_frag_imag[i], dft_matrix_imag_1 + WMMA_K * i, unit); // b
        wmma::mma_sync(k_real, a_frag_imag[i], b_frag_imag[i], k_real);               // bd

        wmma::load_matrix_sync(a_frag_real[i], dft_matrix_real_1 + WMMA_K * i, unit); // a
        wmma::mma_sync(k_imag, a_frag_real[i], b_frag_imag[i], k_imag);               // ad
    }
#pragma unroll
    for (int i = 0; i < k_real.num_elements; i++)
    {
        k_real.x[i] = -k_real.x[i];
    }
#pragma unroll
    for (size_t i = 0; i < 2; i++)
    {
        wmma::load_matrix_sync(b_frag_real[i], input + input_idx_real + i * WMMA_K * ACTUAL_WIDTH, ACTUAL_WIDTH); // c

        wmma::mma_sync(k_imag, a_frag_imag[i], b_frag_real[i], k_imag); // bc
        wmma::mma_sync(k_real, a_frag_real[i], b_frag_real[i], k_real); // ac
    }

    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------

    wmma::fill_fragment(fft_real, 0.0);
    wmma::fill_fragment(fft_imag, 0.0);
#pragma unroll
    for (size_t i = 0; i < 2; i++)
    {
        wmma::load_matrix_sync(b_frag_imag[i], dft_matrix_imag_2 + i * WMMA_K * unit, unit); // d

        a_frag_imag[i].x[0] = k_imag.x[i];                                  // b
        wmma::mma_sync(fft_real, a_frag_imag[i], b_frag_imag[i], fft_real); // bd

        a_frag_real[i].x[0] = k_real.x[i];                                  // a
        wmma::mma_sync(fft_imag, a_frag_real[i], b_frag_imag[i], fft_imag); // ad

        wmma::load_matrix_sync(b_frag_real[i], dft_matrix_real_2 + i * WMMA_K * unit, unit); // c
        wmma::mma_sync(fft_imag, a_frag_real[i], b_frag_real[i], fft_imag);                  // bc
    }
#pragma unroll
    for (int i = 0; i < fft_real.num_elements; i++)
    {
        fft_real.x[i] = -fft_real.x[i];
    }
#pragma unroll
    for (size_t i = 0; i < 2; i++)
    {
        wmma::mma_sync(fft_real, a_frag_real[i], b_frag_real[i], fft_real); // ac
    }

    // --------------------------------------------------------------------------------------------------------------
    // -------------------------------kernel multiplication----------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------

    // element-wise complex multiplication between fft & ifft
    wmma::load_matrix_sync(k_real, kernel_fft_real, unit, wmma::mem_row_major);
    wmma::load_matrix_sync(k_imag, kernel_fft_imag, unit, wmma::mem_row_major);
#pragma unroll
    for (int i = 0; i < k_real.num_elements; i++)
    {
        complexMul(fft_real.x[i], fft_imag.x[i], k_real.x[i], k_imag.x[i], &k_real.x[i], &k_imag.x[i]);
    }

    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------

    /**
     * @brief i f h t
     *
     */
    // real_shared @ idht_matrix_col, 8 * 8（7） @  8 * 8
    wmma::fill_fragment(fft_real, 0.0);
    wmma::fill_fragment(fft_imag, 0.0);

#pragma unroll
    for (size_t i = 0; i < 2; i++)
    {
        b_frag_imag[i].x[0] = -b_frag_imag[i].x[0]; // d

        a_frag_imag[i].x[0] = k_imag.x[i];                                  // b
        wmma::mma_sync(fft_real, a_frag_imag[i], b_frag_imag[i], fft_real); // bd
        wmma::mma_sync(fft_imag, a_frag_imag[i], b_frag_real[i], fft_imag); // bc

        a_frag_real[i].x[0] = k_real.x[i];                                  // a
        wmma::mma_sync(fft_imag, a_frag_real[i], b_frag_imag[i], fft_imag); // ad
    }
#pragma unroll
    for (int i = 0; i < fft_real.num_elements; i++)
    {
        fft_real.x[i] = -fft_real.x[i];
    }
#pragma unroll
    for (size_t i = 0; i < 2; i++)
    {
        wmma::mma_sync(fft_real, a_frag_real[i], b_frag_real[i], fft_real); // ac
    }

    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------

    wmma::store_matrix_sync(real_shared, fft_real, band_unit, wmma::mem_row_major);
    wmma::store_matrix_sync(imag_shared, fft_imag, band_unit, wmma::mem_row_major);

    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------

    wmma::fill_fragment(fft_real, 0.0);
    wmma::fill_fragment(fft_imag, 0.0);

    // 8 * 8 @ 8 * 8（7）
    // idht_matrix_row @ real_shared
#pragma unroll
    for (size_t i = 0; i < 2; i++)
    {
        wmma::load_matrix_sync(b_frag_imag[i], imag_shared + i * WMMA_K * band_unit, band_unit); // d

        wmma::load_matrix_sync(a_frag_imag[i], dft_matrix_imag_1 + WMMA_K * i, unit); // b
        wmma::mma_sync(fft_real, a_frag_imag[i], b_frag_imag[i], fft_real);           // bd

        wmma::load_matrix_sync(a_frag_real[i], dft_matrix_real_1 + WMMA_K * i, unit); // a
        wmma::mma_sync(fft_imag, a_frag_real[i], b_frag_imag[i], fft_imag);           // ad

        wmma::load_matrix_sync(b_frag_real[i], real_shared + i * WMMA_K * band_unit, band_unit); // c
        wmma::mma_sync(fft_imag, a_frag_imag[i], b_frag_real[i], fft_imag);                      // bc
    }
#pragma unroll
    for (int i = 0; i < fft_real.num_elements; i++)
    {
        fft_real.x[i] = -fft_real.x[i];
    }
#pragma unroll
    for (size_t i = 0; i < 2; i++)
    {
        wmma::mma_sync(fft_real, a_frag_real[i], b_frag_real[i], fft_real); // ac
    }

    wmma::store_matrix_sync(real_shared, fft_real, unit, wmma::mem_row_major);
    wmma::store_matrix_sync(imag_shared, fft_imag, unit, wmma::mem_row_major);

    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------

    // const int fft_id_x = blockIdx.x;
    // const int fft_id_y = blockIdx.y;

    const int output_idx_x = blockIdx.y * sub_input_width;
    const int output_idx_y = blockIdx.x * 2 * sub_input_width;

    for (size_t i = laneId; i < rfft_size; i += WARP_SIZE)
    {
        const int row = i / unit;
        const int col = i % unit;

        const int idx_x = (output_idx_x + row);
        const int idx_y_real = (output_idx_y + col);
        const int idx_y_imag = (output_idx_y + sub_input_width + col);

        // if (row < 2 || col < 2 || row >= sub_input_width - 2 || col >= sub_input_width - 2)
        // {
        atomicAdd(output + idx_x * INPUT_WIDTH + idx_y_real, real_shared[i]);
        atomicAdd(output + idx_x * INPUT_WIDTH + idx_y_imag, imag_shared[i]);
        // }
        // else
        // {
        //     output[idx_x * INPUT_WIDTH + idx_y_real] = real_shared[i];
        //     output[idx_x * INPUT_WIDTH + idx_y_imag] = imag_shared[i];
        // }
    }

    // if ((fft_id_x != gridDim.x - 1) && (fft_id_y != gridDim.y - 1))
    // {
    // for (size_t i = laneId; i < rfft_size; i += WARP_SIZE)
    // {
    //     const int row = i / unit;
    //     const int col = i % unit;

    //     const int idx_x = (output_idx_x + row);
    //     const int idx_y_real = (output_idx_y + col);
    //     const int idx_y_imag = (output_idx_y + sub_input_width + col);

    //     atomicAdd(output + idx_x * INPUT_WIDTH + idx_y_real, real_shared[i]);
    //     atomicAdd(output + idx_x * INPUT_WIDTH + idx_y_imag, imag_shared[i]);
    // }
    // }
    // else if ((fft_id_x == gridDim.x - 1) && (fft_id_y != gridDim.y - 1))
    // {
    //     for (size_t i = laneId; i < rfft_size; i += WARP_SIZE)
    //     {
    //         const int row = i / unit;
    //         const int col = i % unit;

    //         const int idx_x = (output_idx_x + row);
    //         const int idx_y_real = (output_idx_y + col);
    //         const int idx_y_imag =
    //             (col >= sub_input_width) ? (col - sub_input_width) : (output_idx_y + sub_input_width + col);

    //         atomicAdd(output + idx_x * INPUT_WIDTH + idx_y_real, real_shared[i]);
    //         atomicAdd(output + idx_x * INPUT_WIDTH + idx_y_imag, imag_shared[i]);
    //     }
    // }
    // else if ((fft_id_x != gridDim.x - 1) && (fft_id_y == gridDim.y - 1))
    // {
    //     for (size_t i = laneId; i < rfft_size; i += WARP_SIZE)
    //     {
    //         const int row = i / unit;
    //         const int col = i % unit;

    //         const int idx_x = (row >= sub_input_width) ? (row - sub_input_width) : (output_idx_x + row);
    //         const int idx_y_real = (output_idx_y + col);
    //         const int idx_y_imag = (output_idx_y + sub_input_width + col);

    //         atomicAdd(output + idx_x * INPUT_WIDTH + idx_y_real, real_shared[i]);
    //         atomicAdd(output + idx_x * INPUT_WIDTH + idx_y_imag, imag_shared[i]);
    //     }
    // }
    // else if ((fft_id_x == gridDim.x - 1) && (fft_id_y == gridDim.y - 1))
    // {
    //     for (size_t i = laneId; i < rfft_size; i += WARP_SIZE)
    //     {
    //         const int row = i / unit;
    //         const int col = i % unit;

    //         const int idx_x = (row >= sub_input_width) ? (row - sub_input_width) : (output_idx_x + row);
    //         const int idx_y_real = (output_idx_y + col);
    //         const int idx_y_imag =
    //             (col >= sub_input_width) ? (col - sub_input_width) : (output_idx_y + sub_input_width + col);

    //         atomicAdd(output + idx_x * INPUT_WIDTH + idx_y_real, real_shared[i]);
    //         atomicAdd(output + idx_x * INPUT_WIDTH + idx_y_imag, imag_shared[i]);
    //     }
    // }
}